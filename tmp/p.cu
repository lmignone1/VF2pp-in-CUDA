#include <stdio.h>
#include <hip/hip_runtime.h>

// Definizione della struttura dati state
typedef struct {
    float *vector1;
    float *vector2;
    float *vector3;
    int** vector4;
} state;

// Funzione per controllare eventuali errori CUDA
void checkCudaError(hipError_t err, const char *msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "Errore CUDA: %s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// Kernel CUDA per riempire i vettori
__global__ void fillVectors(float *vector1, float *vector2, float *vector3, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        vector1[idx] = idx * 1.0f;
        vector2[idx] = idx * 2.0f;
        vector3[idx] = idx * 3.0f;
    }
}

__global__ void fillVector4(int* v, int N, int value) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        v[idx] = value;
    }
}

int main() {
    // Dimensione dei vettori
    int N = 100;
    int size = N * sizeof(float);

    // Allocazione dinamica della struttura state
    state *s;
    s = (state *)malloc(sizeof(state));
    if (s == NULL) {
        fprintf(stderr, "Errore di allocazione della memoria per la struttura state\n");
        return EXIT_FAILURE;
    }

    // Allocazione dei vettori su device (GPU)
    checkCudaError(hipMalloc((void**)&(s->vector1), size), "Allocazione vector1");
    checkCudaError(hipMalloc((void**)&(s->vector2), size), "Allocazione vector2");
    checkCudaError(hipMalloc((void**)&(s->vector3), size), "Allocazione vector3");

    s->vector4 = (int**)malloc(3 * sizeof(int*));
    for (int i = 0; i < 3; i++) {
        checkCudaError(hipMalloc((void**)&(s->vector4[i]), 10*sizeof(int)), "Allocazione vector4");
    }

    // Assicurarsi che l'allocazione sia avvenuta correttamente
    if (s->vector1 == NULL || s->vector2 == NULL || s->vector3 == NULL) {
        fprintf(stderr, "Errore di allocazione della memoria per i vettori\n");
        return EXIT_FAILURE;
    }

    for(int i = 0; i < 3; i++) {
        int* v = s->vector4[i];
        hipMemset(v, 0, 10*sizeof(int));
    }

    // Lanciare il kernel per riempire i vettori
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    fillVectors<<<blocksPerGrid, threadsPerBlock>>>(s->vector1, s->vector2, s->vector3, N);
    checkCudaError(hipGetLastError(), "Lancio del kernel fillVectors");

    for(int i = 0; i < 3; i++) {
        int* v = s->vector4[i];
        fillVector4<<<(10 + threadsPerBlock - 1) / threadsPerBlock, threadsPerBlock>>>(v, 10, i+1);
    }
   
    checkCudaError(hipDeviceSynchronize(), "Sincronizzazione del dispositivo");

    // Copia i dati dai vettori di dispositivo alla memoria host per la verifica
    float *h_vector1 = (float *)malloc(size);
    float *h_vector2 = (float *)malloc(size);
    float *h_vector3 = (float *)malloc(size);
    int* h_vector4 = (int*)malloc(10*sizeof(int));

    if (h_vector1 == NULL || h_vector2 == NULL || h_vector3 == NULL) {
        fprintf(stderr, "Errore di allocazione della memoria per i vettori host\n");
        return EXIT_FAILURE;
    }

    checkCudaError(hipMemcpy(h_vector1, s->vector1, size, hipMemcpyDeviceToHost), "Copia vector1 da device a host");
    checkCudaError(hipMemcpy(h_vector2, s->vector2, size, hipMemcpyDeviceToHost), "Copia vector2 da device a host");
    checkCudaError(hipMemcpy(h_vector3, s->vector3, size, hipMemcpyDeviceToHost), "Copia vector3 da device a host");

    for(int i = 0; i < 3; i++) {
        int* v = s->vector4[i];
        checkCudaError(hipMemcpy(h_vector4, v, 10*sizeof(int), hipMemcpyDeviceToHost), "Copia vector4 da device a host");
        printf("Primi 10 elementi di vector4:\n");
        for (int i = 0; i < 10; i++) {
            printf("%d ", h_vector4[i]);
        }
        printf("\n");
    }

    // Stampa i primi 10 elementi di ogni vettore per la verifica
    printf("Primi 10 elementi di vector1:\n");
    for (int i = 0; i < 10; i++) {
        printf("%f ", h_vector1[i]);
    }
    printf("\n");

    printf("Primi 10 elementi di vector2:\n");
    for (int i = 0; i < 10; i++) {
        printf("%f ", h_vector2[i]);
    }
    printf("\n");

    printf("Primi 10 elementi di vector3:\n");
    for (int i = 0; i < 10; i++) {
        printf("%f ", h_vector3[i]);
    }
    printf("\n");

    // Deallocazione della memoria su device
    checkCudaError(hipFree(s->vector1), "Deallocazione vector1");
    checkCudaError(hipFree(s->vector2), "Deallocazione vector2");
    checkCudaError(hipFree(s->vector3), "Deallocazione vector3");

    // Deallocazione della memoria della struttura state
    free(s);
    // Deallocazione della memoria host
    free(h_vector1);
    free(h_vector2);
    free(h_vector3);

    return 0;
}
