#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "graph_list.h"

// Dichiarazione della funzione esterna
extern "C" Node *createNode(int vertex);
// Kernel CUDA semplice
__global__ void add(int *a, int *b, int *c) {
    int index = threadIdx.x;
    c[index] = a[index] + b[index];
}

int main(void) {
    int a_host[5] = {1, 2, 3, 4, 5};
    int b_host[5] = {10, 20, 30, 40, 50};
    int c_host[5];

    int *a_device, *b_device, *c_device;
    int size = 5 * sizeof(int);

    // Allocazione della memoria sul dispositivo
    hipMalloc((void **)&a_device, size);
    hipMalloc((void **)&b_device, size);
    hipMalloc((void **)&c_device, size);

    // Copia dei dati dalla memoria host alla memoria device
    hipMemcpy(a_device, a_host, size, hipMemcpyHostToDevice);
    hipMemcpy(b_device, b_host, size, hipMemcpyHostToDevice);

    // Lancio del kernel CUDA
    add<<<1, 5>>>(a_device, b_device, c_device);

    // Copia dei risultati dalla memoria device alla memoria host
    hipMemcpy(c_host, c_device, size, hipMemcpyDeviceToHost);

    // Stampa dei risultati
    printf("Risultati dell'addizione nel kernel CUDA:\n");
    for (int i = 0; i < 5; i++) {
        printf("c_host[%d] = %d\n", i, c_host[i]);
    }

    // Utilizzo della funzione esterna
    Node *node = createNode(10);
    printf("Node: %d\n", node->vertex);

    // Libera la memoria allocata sul dispositivo
    hipFree(a_device);
    hipFree(b_device);
    hipFree(c_device);

    return 0;
}
