#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>
#include <limits.h>
#include <string.h>

#define FILENAME_QUERY "data/graph_query_10000.csv"
#define FILENAME_TARGET "data/graph_target_10000.csv"
#define STREAMS 6
#define LABELS 10
#define INF 99999
#define MAXCONSTMEM 16000

int blockSize;
__constant__ int constMem[MAXCONSTMEM];
__constant__ int constMemVar;

#define CUDA_CHECK_ERROR(err)           \
    if (err != hipSuccess) {            \
        printf("CUDA error: %s\n", hipGetErrorString(err)); \
        printf("Error in file: %s, line: %i\n", __FILE__, __LINE__); \
        exit(EXIT_FAILURE);              \
    }

/***** STRUCTS *****/
typedef struct {
    int* matrix;
    int numVertices;
    int* nodesToLabel;
    int** labelToNodes;
    int* labelsCardinalities;
    int* degrees;
} Graph;

typedef struct {
    int *mapping1;  // mapping from query to target
    int *mapping2;  // mapping from target to query
    int *T1;        // Ti contains uncovered neighbors of covered nodes from Gi, i.e. nodes that are not in the mapping, but are neighbors of nodes that are.
    int *T2;
    int* T1_out;     //Ti_out contains all the nodes from Gi, that are neither in the mapping nor in Ti. Cioe nodi che non sono in mapping e non sono vicini di nodi coperti
    int* T2_out;
} State;

typedef struct {
    int vertex;
    int* candidates;
    int sizeCandidates;
    int candidateIndex;
} Info;

typedef struct StackNode {
    Info* info;
    struct StackNode* next;
} StackNode;

/***** GRAPH PROTOTYPES *****/
void initGraphGPU(Graph*);
Graph* createGraph();
void addEdge(Graph*, int, int);
Graph* readGraph(char*);
void printGraph(Graph*);
void freeGraph(Graph*);
void setLabel(Graph*, int, int);

Graph* graphGPU(Graph*);
void freeGraphGPU(Graph*);

/***** STATE PROTOTYPES *****/
State* createStateGPU(Graph*, Graph*, hipStream_t*);
void freeStateGPU(State*);
void printState(State*, int);
void updateStateGPU(Graph*, Graph*, State*, int, int, hipStream_t*);
void restoreStateGPU(Graph*, Graph*, State*, int, int, hipStream_t*);

/***** VF2++ PROTOTYPES *****/
void vf2ppGPU(Graph*, Graph*, State*, Graph*, Graph*, hipStream_t*);
bool checkGraphPropertiesGPU(Graph*, Graph*, Graph*, Graph*, hipStream_t*);
int compare(const void*, const void*);
int* orderingGPU(Graph*, Graph*, hipStream_t*, Graph*);
void findRootGPU(Graph*, int*, int*, int*, int*, hipStream_t*);
void processDepthGPU(Graph*, int*, int*, int*, int*, int*, int*, int*, int*, int*, int*, int*, int*, hipStream_t*, Graph*);
int* findCandidatesGPU(Graph*, Graph*, State*, int, int*, Graph*, Graph*, hipStream_t*);
bool cutISOGPU(Graph*, Graph*, State*, int, int, Graph*, Graph*, hipStream_t*);

/***** STACK PROTOTYPES *****/
Info* createInfo(int* candidates, int sizeCandidates, int vertex);
StackNode* createStackNode(Info*);
void push(StackNode**, Info*);
Info* pop(StackNode**);
bool isStackEmpty(StackNode*);
void freeStack(StackNode*);
void printStack(StackNode*);
void printInfo(Info*);
void freeInfo(Info*);
StackNode* createStack();
Info* peek(StackNode**);

int main() {
    blockSize = 256;

    Graph* h_g1 = readGraph(FILENAME_QUERY);
    Graph* h_g2 = readGraph(FILENAME_TARGET);
    Graph* d_g1 = graphGPU(h_g1);
    Graph* d_g2 = graphGPU(h_g2);

    hipStream_t streams[STREAMS];

    for(int i = 0; i < STREAMS; i++) {
        hipStreamCreate(&streams[i]);
    }

    State* d_state = createStateGPU(d_g1, d_g2, streams);

    vf2ppGPU(d_g1, d_g2, d_state, h_g1, h_g2, streams);

    int* mapping1 = (int*)malloc(d_g1->numVertices * sizeof(int));

    if(mapping1 == NULL) {
        printf("Error allocating memory in main\n");
        exit(EXIT_FAILURE);
    }

    CUDA_CHECK_ERROR(hipMemcpy(mapping1, d_state->mapping1, d_g1->numVertices * sizeof(int), hipMemcpyDeviceToHost));

    printf("Mapping\n");
    for(int i = 0; i < d_g1->numVertices; i++) {
        printf("%d -> %d\n", i, mapping1[i]);
    }

    for(int i = 0; i < STREAMS; i++) {
        hipStreamDestroy(streams[i]);
    }

    freeGraph(h_g1);
    freeGraph(h_g2);
    freeGraphGPU(d_g1);
    freeGraphGPU(d_g2);

    freeStateGPU(d_state);

    return EXIT_SUCCESS;
}

__global__ void initArrayKernel(int* d_array, int size, int value) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        d_array[idx] = value;
    }
}

__global__ void initMatrixKernel(int* d_matrix, int V, int value) {
    int col = threadIdx.y + blockIdx.y * blockDim.y;
    int row = threadIdx.x + blockIdx.x * blockDim.x;

    if (row < V && col < V) {
        d_matrix[row * V + col] = value;
    }
}

/***** GRAPH FUNCTIONS *****/
void initGraphGPU(Graph* g) {
    g->matrix = (int*)malloc(g->numVertices * g->numVertices * sizeof(int));
    g->nodesToLabel = (int*)malloc(g->numVertices * sizeof(int));
    g->labelsCardinalities = (int*)malloc(LABELS * sizeof(int));
    g->labelToNodes = (int**)malloc(LABELS * sizeof(int*));
    g->degrees = (int*)malloc(g->numVertices * sizeof(int));

    if (g->nodesToLabel == NULL || g->labelsCardinalities == NULL || g->labelToNodes == NULL || g->degrees == NULL || g->matrix == NULL) {
        printf("Error allocating memory in initGraph\n");
        exit(EXIT_FAILURE);
    }

    int *d_nodesToLabel, *d_degrees, *d_matrix;

    hipStream_t stream1, stream2, stream3;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);

    CUDA_CHECK_ERROR(hipMalloc((void**)&d_nodesToLabel, g->numVertices * sizeof(int)));
    CUDA_CHECK_ERROR(hipMalloc((void**)&d_degrees, g->numVertices * sizeof(int)));
    CUDA_CHECK_ERROR(hipMalloc((void**)&d_matrix, g->numVertices * g->numVertices * sizeof(int)));

    int gridSize = (g->numVertices + blockSize - 1) / blockSize;

    initArrayKernel<<<gridSize, blockSize, 0, stream1>>>(d_nodesToLabel, g->numVertices, -1);
    initArrayKernel<<<gridSize, blockSize, 0, stream2>>>(d_degrees, g->numVertices, 0);

    int gridSizeX = (g->numVertices + blockSize - 1) / blockSize;
    int gridSizeY = (g->numVertices + blockSize - 1) / blockSize;
    dim3 gridSizeM(gridSizeX, gridSizeY);

    dim3 blockSizeM(blockSize, blockSize);
    initMatrixKernel<<<gridSizeM, blockSizeM, 0, stream3>>>(d_matrix, g->numVertices, 0);

    for (int label = 0; label < LABELS; label++) {
        g->labelsCardinalities[label] = 0;
        g->labelToNodes[label] = (int*)malloc(g->numVertices * sizeof(int));
    }

    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);
    hipStreamSynchronize(stream3);

    CUDA_CHECK_ERROR(hipMemcpy(g->nodesToLabel, d_nodesToLabel, g->numVertices * sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CHECK_ERROR(hipMemcpy(g->degrees, d_degrees, g->numVertices * sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CHECK_ERROR(hipMemcpy(g->matrix, d_matrix, g->numVertices * g->numVertices * sizeof(int), hipMemcpyDeviceToHost));

    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipStreamDestroy(stream3);

    hipFree(d_nodesToLabel);
    hipFree(d_degrees);
    hipFree(d_matrix);
}

void setLabel(Graph* g, int node, int label) {
    if (g->nodesToLabel[node] == -1) {
        g->nodesToLabel[node] = label;
        g->labelsCardinalities[label]++;
        g->labelToNodes[label][g->labelsCardinalities[label] - 1] = node;
    }
}

void addEdge(Graph* g, int src, int target) {
    g->matrix[src * g->numVertices + target] = 1;
    g->matrix[target * g->numVertices + src] = 1;
    g->degrees[src]++;
    g->degrees[target]++;
}

Graph* createGraph() {
    Graph* g = (Graph*)malloc(sizeof(Graph));

    if (g == NULL) {
        printf("Error allocating memory in createGraph\n");
        exit(EXIT_FAILURE);
    }

    g->matrix = NULL;
    g->numVertices = 0;
    g->nodesToLabel = NULL;
    g->labelsCardinalities = NULL;
    g->degrees = NULL;
    g->labelToNodes = NULL;
    return g;
}

Graph* readGraph(char* path) {
    int src, target, srcLabel, targetLabel;
    Graph* g = createGraph();

    FILE* f = fopen(path, "r");
    if (f == NULL) {
        printf("Error opening file\n");
        exit(EXIT_FAILURE);
    }

    char line[128];
    fgets(line, sizeof(line), f);
    sscanf(line, "%*s%*s%*s%d", &g->numVertices);
    fgets(line, sizeof(line), f); // skip the header

    initGraphGPU(g);

    while (fgets(line, sizeof(line), f)) {
        sscanf(line, "%d,%d,%d,%d", &src, &target, &srcLabel, &targetLabel);
        addEdge(g, src, target);
        setLabel(g, src, srcLabel);
        setLabel(g, target, targetLabel);
    }

    fclose(f);

    for(int label = 0; label < LABELS; label++) {
        g->labelToNodes[label] = (int*)realloc(g->labelToNodes[label], g->labelsCardinalities[label] * sizeof(int));
    }

    return g;
}

void printGraph(Graph* g) {
    for (int i = 0; i < g->numVertices; i++) {
        for (int j = 0; j < g->numVertices; j++) {
            printf("%d ", g->matrix[i * g->numVertices + j]);
        }
        printf("\tVertex %d, label %d, degree %d\n", i, g->nodesToLabel[i], g->degrees[i]);
    }

    printf("\nCardinalities\n");
    for (int i = 0; i < LABELS; i++) {
        printf("Label %d: %d\n", i, g->labelsCardinalities[i]);
    }

    for(int i = 0; i < LABELS; i++) {
       printf("\nLabel %d\n", i);
       for(int j = 0; j < g->labelsCardinalities[i]; j++) {
           printf("%d ", g->labelToNodes[i][j]);
       }
    }
}

void freeGraph(Graph* g) {
    for(int i = 0; i < LABELS; i++) {
        free(g->labelToNodes[i]);
    }
    free(g->labelToNodes);
    free(g->matrix);
    free(g->nodesToLabel);
    free(g->labelsCardinalities);
    free(g->degrees);
    free(g);
    g = NULL;
}

Graph* graphGPU(Graph* h_g) {
    Graph* d_g = createGraph();

    size_t sizeMatrix = h_g->numVertices * h_g->numVertices * sizeof(int);
    size_t size1 = h_g->numVertices * sizeof(int);
    size_t size2 = LABELS * sizeof(int);

    d_g->numVertices = h_g->numVertices;
    d_g->labelToNodes = (int**)malloc(LABELS * sizeof(int*));   // it is a vector on host wich contains pointers to vectors on device

    CUDA_CHECK_ERROR(hipMalloc((void**)&d_g->matrix, sizeMatrix));
    CUDA_CHECK_ERROR(hipMalloc((void**)&d_g->nodesToLabel, size1));
    CUDA_CHECK_ERROR(hipMalloc((void**)&d_g->labelsCardinalities, size2));
    CUDA_CHECK_ERROR(hipMalloc((void**)&d_g->degrees, size1));

    for(int label = 0; label < LABELS; label++) {
        CUDA_CHECK_ERROR(hipMalloc((void**)&d_g->labelToNodes[label], h_g->labelsCardinalities[label] * sizeof(int))); // each vector on device has a size equal to the cardinality of the label
    }

    CUDA_CHECK_ERROR(hipMemcpy(d_g->matrix, h_g->matrix, sizeMatrix, hipMemcpyHostToDevice));
    CUDA_CHECK_ERROR(hipMemcpy(d_g->nodesToLabel, h_g->nodesToLabel, size1, hipMemcpyHostToDevice));
    CUDA_CHECK_ERROR(hipMemcpy(d_g->labelsCardinalities, h_g->labelsCardinalities, size2, hipMemcpyHostToDevice));
    CUDA_CHECK_ERROR(hipMemcpy(d_g->degrees, h_g->degrees, size1, hipMemcpyHostToDevice));

    for(int label = 0; label < LABELS; label++) {
        CUDA_CHECK_ERROR(hipMemcpy(d_g->labelToNodes[label], h_g->labelToNodes[label], h_g->labelsCardinalities[label] * sizeof(int), hipMemcpyHostToDevice));
    }

    return d_g;
}

void freeGraphGPU(Graph* g) {
    for(int label = 0; label < LABELS; label++) {
        hipFree(g->labelToNodes[label]);
    }
    free(g->labelToNodes);
    hipFree(g->matrix);
    hipFree(g->nodesToLabel);
    hipFree(g->labelsCardinalities);
    hipFree(g->degrees);
    free(g);
    g = NULL;
}

/***** STATE FUNCTIONS *****/
State* createStateGPU(Graph* g1, Graph* g2, hipStream_t* streams) {
    State* s = (State*)malloc(sizeof(State));

    if (s == NULL) {
        printf("Error allocating memory in createStateGPU\n");
        exit(EXIT_FAILURE);
    }

    size_t size1 = g1->numVertices * sizeof(int);
    size_t size2 = g2->numVertices * sizeof(int);

    CUDA_CHECK_ERROR(hipMalloc((void**)&s->mapping1, size1));
    CUDA_CHECK_ERROR(hipMalloc((void**)&s->mapping2, size2));
    CUDA_CHECK_ERROR(hipMalloc((void**)&s->T1, size1));
    CUDA_CHECK_ERROR(hipMalloc((void**)&s->T2, size2));
    CUDA_CHECK_ERROR(hipMalloc((void**)&s->T1_out, size1));
    CUDA_CHECK_ERROR(hipMalloc((void**)&s->T2_out, size2));

    int gridSize1 = (g1->numVertices + blockSize - 1) / blockSize;
    int gridSize2 = (g2->numVertices + blockSize - 1) / blockSize;

    hipStream_t stream1 = streams[0];
    hipStream_t stream2 = streams[1];
    hipStream_t stream3 = streams[2];

    initArrayKernel<<<gridSize1, blockSize, 0, stream1>>>(s->mapping1, g1->numVertices, -1);
    initArrayKernel<<<gridSize1, blockSize, 0, stream2>>>(s->T1, g1->numVertices, -1);
    initArrayKernel<<<gridSize1, blockSize, 0, stream3>>>(s->T1_out, g1->numVertices, 1);

    initArrayKernel<<<gridSize2, blockSize, 0, stream1>>>(s->mapping2, g2->numVertices, -1);
    initArrayKernel<<<gridSize2, blockSize, 0, stream2>>>(s->T2, g2->numVertices, -1);
    initArrayKernel<<<gridSize2, blockSize, 0, stream3>>>(s->T2_out, g2->numVertices, 1);

    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);
    hipStreamSynchronize(stream3);

    return s;
}

void freeStateGPU(State* s) {
    hipFree(s->mapping1);
    hipFree(s->mapping2);
    hipFree(s->T1);
    hipFree(s->T2);
    hipFree(s->T1_out);
    hipFree(s->T2_out);
    free(s);
    s = NULL;
}

void printState(State* s, int numVertices) {
    printf("Mapping 1\n");
    for (int i = 0; i < numVertices; i++) {
        printf("%d ", s->mapping1[i]);
    }

    printf("\nMapping 2\n");
    for (int i = 0; i < numVertices; i++) {
        printf("%d ", s->mapping2[i]);
    }

    printf("\nT1\n");
    for (int i = 0; i < numVertices; i++) {
        if(s->T1[i] != -1) {
            printf("%d ", i);
        }
        // printf("%d ", s->T1[i]);
    }

    printf("\nT2\n");
    for (int i = 0; i < numVertices; i++) {
        if(s->T2[i] != -1) {
            printf("%d ", i);
        }
        // printf("%d ", s->T2[i]);
    }

    printf("\nT1_out\n");
    for (int i = 0; i < numVertices; i++) {
        if(s->T1_out[i] != -1) {
            printf("%d ", i);
        }
        // printf("%d ", s->T1_out[i]);
    }

    printf("\nT2_out\n");
    for (int i = 0; i < numVertices; i++) {
        if(s->T2_out[i] != -1) {
            printf("%d ", i);
        }
        // printf("%d ", s->T2_out[i]);
    }
}

__global__ void updateStateKernel(int* matrix, int V, int* mapping, int* T, int* T_out, int node1, int node2) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if( idx < V) {
        if(idx == node1) {
            mapping[idx] = node2;
        }

        __syncthreads();

        if(matrix[node1 * V + idx] == 1 && mapping[idx] == -1) {
            T[idx] = 1;
            T_out[idx] = -1;
        }

        __syncthreads();

        if(idx == node1) {
            T[idx] = -1;
            T_out[idx] = -1;
        }
    }
}

void updateStateGPU(Graph* d_g1, Graph* d_g2, State* d_state, int node, int candidate, hipStream_t* streams) {
    hipStream_t stream1 = streams[0];
    hipStream_t stream2 = streams[1];

    int gridSize = (d_g1->numVertices + blockSize - 1) / blockSize;
    updateStateKernel<<<gridSize, blockSize, 0, stream1>>>(d_g1->matrix, d_g1->numVertices, d_state->mapping1, d_state->T1, d_state->T1_out, node, candidate);

    gridSize = (d_g2->numVertices + blockSize - 1) / blockSize;
    updateStateKernel<<<gridSize, blockSize, 0, stream2>>>(d_g2->matrix, d_g2->numVertices, d_state->mapping2, d_state->T2, d_state->T2_out, candidate, node);
}

__global__ void restoreStateKernel(int* matrix, int V, int node, int* T, int* T_out, int offset, int usage, int* mapping) {  // offset introduced because of the two streams
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= V) {
        return;
    }

    int isAdded = 0;

    if(matrix[node * V + idx] == 1) {

        if(usage == 1) {

            if(constMem[offset + idx] != -1) {  // constMem contains mapping1 if offset is 0, mapping2 if offset is V
                atomicExch(&T[node], 1);
                isAdded = 1;
            }
            else {
                int hasCoveredNeighbor = 0;
                for(int adjVertex2 = 0; adjVertex2 < V; adjVertex2++) {
                    if(matrix[idx * V + adjVertex2] == 1 && constMem[offset + adjVertex2] != -1) {   // constMem contains mapping1 if offset is 0, mapping2 if offset is V
                        hasCoveredNeighbor = 1;
                        break;
                    }
                }

                if(hasCoveredNeighbor == 0) {
                    T[idx] = -1;
                    T_out[idx] = 1;
                }
            }

        } else {

            if(mapping[idx] != -1) { 
                atomicExch(&T[node], 1);
                isAdded = 1;
            }
            else {
                int hasCoveredNeighbor = 0;
                for(int adjVertex2 = 0; adjVertex2 < V; adjVertex2++) {
                    if(matrix[idx * V + adjVertex2] == 1 && mapping[adjVertex2] != -1) {   
                        hasCoveredNeighbor = 1;
                        break;
                    }
                }

                if(hasCoveredNeighbor == 0) {
                    T[idx] = -1;
                    T_out[idx] = 1;
                }
            }

        }
    }

    if(isAdded == 0) {
        atomicExch(&T_out[node], 1);
    }
}

void restoreStateGPU(Graph* d_g1, Graph* d_g2, State* d_state, int node, int candidate, hipStream_t* streams) {
    hipStream_t stream1 = streams[0];
    hipStream_t stream2 = streams[1];

    size_t size1 = d_g1->numVertices * sizeof(int);
    size_t size2 = d_g2->numVertices * sizeof(int);
    int value = -1;

    int useConstMem = d_g1->numVertices + d_g2->numVertices < MAXCONSTMEM;

    CUDA_CHECK_ERROR(hipMemcpyAsync(d_state->mapping1 + node, &value, sizeof(int), hipMemcpyHostToDevice, stream1));
    CUDA_CHECK_ERROR(hipMemcpyAsync(d_state->mapping2 + candidate, &value, sizeof(int), hipMemcpyHostToDevice, stream2));

    if(useConstMem) {
        CUDA_CHECK_ERROR(hipMemcpyToSymbolAsync(HIP_SYMBOL(constMem), d_state->mapping1, size1, 0, hipMemcpyDeviceToDevice, stream1));
        CUDA_CHECK_ERROR(hipMemcpyToSymbolAsync(HIP_SYMBOL(constMem), d_state->mapping2, size2, size1, hipMemcpyDeviceToDevice, stream2));
    }
    
    int gridSize = (d_g1->numVertices + blockSize - 1) / blockSize;
    restoreStateKernel<<<gridSize, blockSize, 0, stream1>>>(d_g1->matrix, d_g1->numVertices, node, d_state->T1, d_state->T1_out, 0, useConstMem, d_state->mapping1);
    
    gridSize = (d_g2->numVertices + blockSize - 1) / blockSize;
    restoreStateKernel<<<gridSize, blockSize, 0, stream2>>>(d_g2->matrix, d_g2->numVertices, candidate, d_state->T2, d_state->T2_out, d_g1->numVertices, useConstMem, d_state->mapping2);

    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);
}

/***** VF2++ FUNCTIONS *****/
__global__ void equalKernel(int* arr1, int* arr2, int size, int* ret) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size) {
        if (arr1[idx] != arr2[idx]) {
            *ret = 0;
        }
    }
}

int compare(const void* a, const void* b) {
    return (*(int*)a - *(int*)b);
}

bool checkGraphPropertiesGPU(Graph* h_g1, Graph* h_g2, Graph* d_g1, Graph* d_g2, hipStream_t* streams) {
    if (h_g1->numVertices != h_g2->numVertices || h_g1->numVertices == 0 || h_g2->numVertices == 0) {
        return false;
    }

    int h_ret1 = 1, h_ret2 = 1;
    int* d_ret1, *d_ret2;

    hipStream_t stream1 = streams[0];
    hipStream_t stream2 = streams[1];

    // first check: the cardinalities of the labels must be the same
    CUDA_CHECK_ERROR(hipMalloc((void**)&d_ret1, sizeof(int)));
    CUDA_CHECK_ERROR(hipMemcpyAsync(d_ret1, &h_ret1, sizeof(int), hipMemcpyHostToDevice, stream1));

    int gridSize1 = (LABELS + blockSize - 1) / blockSize;
    equalKernel<<<gridSize1, blockSize, 0, stream1>>>(d_g1->labelsCardinalities, d_g2->labelsCardinalities, LABELS, d_ret1);

    // second check: the sequence of the degrees must be the same
    int size = h_g1->numVertices;
    int *d_tmp1, *d_tmp2;

    int* tmp1 = (int*)malloc(size * sizeof(int));
    int* tmp2 = (int*)malloc(size * sizeof(int));

    memcpy(tmp1, h_g1->degrees, size * sizeof(int));
    memcpy(tmp2, h_g2->degrees, size * sizeof(int));
    
    qsort(tmp1, size, sizeof(int), compare);
    qsort(tmp2, size, sizeof(int), compare);

    CUDA_CHECK_ERROR(hipMalloc((void**)&d_tmp1, size * sizeof(int)));
    CUDA_CHECK_ERROR(hipMalloc((void**)&d_tmp2, size * sizeof(int)));
    CUDA_CHECK_ERROR(hipMalloc((void**)&d_ret2, sizeof(int)));

    CUDA_CHECK_ERROR(hipMemcpyAsync(d_tmp1, tmp1, size * sizeof(int), hipMemcpyHostToDevice, stream2));
    CUDA_CHECK_ERROR(hipMemcpyAsync(d_tmp2, tmp2, size * sizeof(int), hipMemcpyHostToDevice, stream2));
    CUDA_CHECK_ERROR(hipMemcpyAsync(d_ret2, &h_ret2, sizeof(int), hipMemcpyHostToDevice, stream2));
   
    int gridSize2 = (size + blockSize - 1) / blockSize;
    equalKernel<<<gridSize2, blockSize, 0, stream2>>>(d_tmp1, d_tmp2, size, d_ret2);

    CUDA_CHECK_ERROR(hipMemcpyAsync(&h_ret1, d_ret1, sizeof(int), hipMemcpyDeviceToHost, stream1));
    CUDA_CHECK_ERROR(hipMemcpyAsync(&h_ret2, d_ret2, sizeof(int), hipMemcpyDeviceToHost, stream2));

    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);

    hipFree(d_ret1);
    hipFree(d_ret2);
    hipFree(d_tmp2);
    hipFree(d_tmp1);
    
    free(tmp1);
    free(tmp2);

    return h_ret1 && h_ret2;
}

__global__ void bfsKernel(int* matrix, int V, int* levels, int* d_done, int depth) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ int s_done;
    extern __shared__ int s_levels[];

    if(threadIdx.x == 0) {
        s_done = 0;
    }

    // it copies the whole levels array in shared memory
    // example: thread 0 loads levels[0] and levels[4] if blockSize is 4
    for(int i = threadIdx.x; i < V; i += blockDim.x) {
      s_levels[i] = levels[i];
    }

    __syncthreads();

    // levels is used as visited too
    if(idx < V && s_levels[idx] == depth) {    // it blocks all thread with size greater than V and all threads not at the current depth

        for(int adjVertex = 0; adjVertex < V; adjVertex++) {
            if(matrix[idx * V + adjVertex] == 1 && s_levels[adjVertex] == -1) {
                atomicExch(&levels[adjVertex], depth + 1);
                s_done = 1;
            }
        }
    }

    __syncthreads();

    if(threadIdx.x == 0) {
        atomicExch(d_done, s_done);
    }
}

__global__ void maxRarityConstMemKernel(int V, int* d_nodesToLabel, int* d_maxRarity, int* d_is_good) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    extern __shared__ int s_data[];

    if(idx < V && d_is_good[idx]) {
        s_data[threadIdx.x] = constMem[d_nodesToLabel[idx]];    // constMem contains labelRarity
    } else {
        s_data[threadIdx.x] = INF;
    }

    __syncthreads();

    // parallel reduction: at each step, the block is halved and each thread computes the min of its value with the value of the other one at distance s in
    // the same block
    for(unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if(threadIdx.x < s) {
            s_data[threadIdx.x] = min(s_data[threadIdx.x], s_data[threadIdx.x + s]);   // At the end of the loop, the min value is in sdata[0]
        }
        __syncthreads();
    }

    // each thread 0 of each block computes the global min
    if(threadIdx.x == 0) {
        atomicMin(d_maxRarity, s_data[0]);
    }
}

__global__ void maxRarityConstMemFilterKernel(int V, int* d_nodesToLabel, int* d_maxRarity, int* d_is_good) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx < V) {
        if(constMem[d_nodesToLabel[idx]] != *d_maxRarity) {     // constMem contains labelRarity
            d_is_good[idx] = 0;
        }
    }
}

__global__ void maxDegreeKernel(int V, int* d_degrees, int* d_maxDegree, int* d_is_good) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    extern __shared__ int s_data[];

    if(idx < V && d_is_good[idx]) {
        s_data[threadIdx.x] = d_degrees[idx];
    }
    else {
        s_data[threadIdx.x] = -INF;
    }

    __syncthreads();

    for(unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if(threadIdx.x < s) {
            s_data[threadIdx.x] = max(s_data[threadIdx.x], s_data[threadIdx.x + s]);
        }
        __syncthreads();
    }

    if(threadIdx.x == 0) {
        atomicMax(d_maxDegree, s_data[0]);
    }
}

__global__ void maxDegreeFilterKernel(int V, int* d_degrees, int* d_maxDegree, int* d_is_good) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx < V) {
        if(d_degrees[idx] != *d_maxDegree) {
            d_is_good[idx] = 0;
        }
    }
}

__global__ void findNodeKernel(int V, int* is_good, int* d_node) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < V && is_good[idx]) {
        atomicExch(d_node, idx);
    }
}

void findRootGPU(Graph* d_g, int* d_root, int* d_is_good, int* d_maxRarity, int* d_maxDegree, hipStream_t* streams) {
    hipStream_t stream1 = streams[0];
    hipStream_t stream2 = streams[1];

    int h_maxRarity = INF, h_maxDegree = -INF;
    int gridSize = (d_g->numVertices + blockSize - 1) / blockSize;
    size_t size2 = LABELS * sizeof(int);

    initArrayKernel<<<gridSize, blockSize, 0, stream1>>>(d_is_good, d_g->numVertices, 1);
    CUDA_CHECK_ERROR(hipMemcpyAsync(d_maxRarity, &h_maxRarity, sizeof(int), hipMemcpyHostToDevice, stream2));
    CUDA_CHECK_ERROR(hipMemcpyAsync(d_maxDegree, &h_maxDegree, sizeof(int), hipMemcpyHostToDevice, stream1));
    CUDA_CHECK_ERROR(hipMemcpyToSymbolAsync(HIP_SYMBOL(constMem), d_g->labelsCardinalities, size2, 0, hipMemcpyDeviceToDevice, stream2)); // constMem contains labelRarity (d_g1->labelsCardinalities)

    size_t sharedMemSize = blockSize * sizeof(int); // each block has a shared memory of size blockSize

    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);

    maxRarityConstMemKernel<<<gridSize, blockSize, sharedMemSize, stream1>>>(d_g->numVertices, d_g->nodesToLabel, d_maxRarity, d_is_good);
    maxRarityConstMemFilterKernel<<<gridSize, blockSize, 0, stream1>>>(d_g->numVertices, d_g->nodesToLabel, d_maxRarity, d_is_good);
    maxDegreeKernel<<<gridSize, blockSize, sharedMemSize, stream1>>>(d_g->numVertices, d_g->degrees, d_maxDegree, d_is_good);
    maxDegreeFilterKernel<<<gridSize, blockSize, 0, stream1>>>(d_g->numVertices, d_g->degrees, d_maxDegree, d_is_good);
    findNodeKernel<<<gridSize, blockSize, 0, stream1>>>(d_g->numVertices, d_is_good, d_root);
}

__global__ void findLevelNodesKernel(int* levels, int depth, int* levelNodes, int V, int* levelSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ int s_levelSize;

    if(threadIdx.x == 0) {
        s_levelSize = 0;
    }

    __syncthreads();

    if(idx < V && levels[idx] == depth) {
        atomicAdd(&s_levelSize, 1);
        levelNodes[idx] = 1;
    }

    __syncthreads();

    if(threadIdx.x == 0) {
        atomicAdd(levelSize, s_levelSize);
    }
}

__global__ void initBfsKernel(int* levels, int V, int* root, int depth) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(idx < V) {
        if(idx == *root) {
            levels[idx] = depth;
        }
        else {
            levels[idx] = -1;
        }
    }
}

// __global__ void printArrayKernel(int* arr, int V) {
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     if( idx == 0)
//       for(int i = 0; i < V; i++)
//         printf("%d ", arr[i]);
// }

// __global__ void printVarKernel(int* d) {
//     printf(" .%d. ", *d);
// }

int* orderingGPU(Graph* d_g1, Graph* d_g2, hipStream_t* streams, Graph* h_g1) {
    // findRootGPU
    int *d_root, *d_is_good, *d_maxRarity, *d_maxDegree;

    // BFS
    int* d_levels, *d_done;
    int* h_done;
    int depth = 0;

    // findLevelNodesKernel
    int *d_levelNodes, *d_levelSize;

    // processDepth
    int *d_V1Unordered, *d_labelRarity, *d_connectivityG1, *d_maxConnectivity;  // d_root is reused as d_nextNode
    int* order = (int*)malloc(d_g1->numVertices * sizeof(int));   // order of the nodes of g1
    int order_index = 0;

    size_t size1 = d_g1->numVertices * sizeof(int);
    size_t size2 = LABELS * sizeof(int);
    int gridSize = (d_g1->numVertices + blockSize - 1) / blockSize;

    hipStream_t stream1 = streams[0];
    hipStream_t stream2 = streams[1];
    hipStream_t stream3 = streams[2];
    hipStream_t stream4 = streams[3];
    
    CUDA_CHECK_ERROR(hipMalloc((void**)&d_root, sizeof(int)));
    CUDA_CHECK_ERROR(hipMalloc((void**)&d_is_good, size1));
    CUDA_CHECK_ERROR(hipMalloc((void**)&d_maxRarity, sizeof(int)));
    CUDA_CHECK_ERROR(hipMalloc((void**)&d_maxDegree, sizeof(int)));
    
    findRootGPU(d_g1, d_root, d_is_good, d_maxRarity, d_maxDegree, streams);

    CUDA_CHECK_ERROR(hipMalloc((void**)&d_levels, size1));
    CUDA_CHECK_ERROR(hipMalloc((void**)&d_done, sizeof(int)));
    CUDA_CHECK_ERROR(hipHostMalloc((void**)&h_done, sizeof(int))); // pinned memory for faster host-device communication

    CUDA_CHECK_ERROR(hipMalloc((void**)&d_levelNodes, size1));
    CUDA_CHECK_ERROR(hipMalloc((void**)&d_levelSize, sizeof(int)));
    
    CUDA_CHECK_ERROR(hipMalloc((void**)&d_V1Unordered, size1));
    CUDA_CHECK_ERROR(hipMalloc((void**)&d_labelRarity, size2));
    CUDA_CHECK_ERROR(hipMalloc((void**)&d_connectivityG1, size1));
    CUDA_CHECK_ERROR(hipMalloc((void**)&d_maxConnectivity, sizeof(int)));

    initArrayKernel<<<gridSize, blockSize, 0, stream2>>>(d_V1Unordered, d_g1->numVertices, -1); // stream2 empty because already sync in findRootGPU
    CUDA_CHECK_ERROR(hipMemcpyAsync(d_labelRarity, d_g1->labelsCardinalities, size2, hipMemcpyDeviceToDevice, stream3));
    CUDA_CHECK_ERROR(hipMemsetAsync(d_connectivityG1, 0, size1, stream4));

    initBfsKernel<<<gridSize, blockSize, 0, stream1>>>(d_levels, d_g1->numVertices, d_root, depth); // executed after findRootGPU because of same stream
    
    size_t sharedMemSize = size1;

    do {
        *h_done = 0;
        CUDA_CHECK_ERROR(hipMemsetAsync(d_done, 0, sizeof(int), stream1));

        bfsKernel<<<gridSize, blockSize, sharedMemSize, stream1>>>(d_g1->matrix, d_g1->numVertices, d_levels, d_done, depth);

        CUDA_CHECK_ERROR(hipMemcpyAsync(h_done, d_done, sizeof(int), hipMemcpyDeviceToHost, stream1));
        depth++;
       
        hipStreamSynchronize(stream1);
    } while(*h_done);
  
    hipStreamSynchronize(stream2); // bfs at stream1 is implicitly synchronized 
    hipStreamSynchronize(stream3);
    hipStreamSynchronize(stream4);
    
    for (int d = 0; d < depth; d++) {
        CUDA_CHECK_ERROR(hipMemsetAsync(d_levelNodes, 0, size1, stream2));
        CUDA_CHECK_ERROR(hipMemsetAsync(d_levelSize, 0, sizeof(int), stream1));

        hipStreamSynchronize(stream2);
        
        findLevelNodesKernel<<<gridSize, blockSize, 0, stream1>>>(d_levels, d, d_levelNodes, d_g1->numVertices, d_levelSize);
        
        processDepthGPU(d_g1, order, &order_index, d_connectivityG1, d_labelRarity, d_V1Unordered, d_levelNodes, d_levelSize, 
                        d_is_good, d_maxRarity, d_maxDegree, d_maxConnectivity, d_root, streams, h_g1);
    }

    // findRootGPU
    hipFree(d_root);
    hipFree(d_is_good);
    hipFree(d_maxRarity);
    hipFree(d_maxDegree);

    //BFS
    hipHostFree(h_done);
    hipFree(d_done);
    hipFree(d_levels);
    
    // findLevelNodesKernel
    hipFree(d_levelNodes);
    hipFree(d_levelSize);

    // processDepth
    hipFree(d_V1Unordered);
    hipFree(d_labelRarity);
    hipFree(d_connectivityG1);
    hipFree(d_maxConnectivity);

    return order;
}

__global__ void maxConnectivityKernel(int* d_connectivityG1, int* d_maxConnectivity, int* d_is_good, int V) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    extern __shared__ int s_data[];

    if(idx < V && d_is_good[idx]) {         // d_is_good already contains the information about the nodes of the current level
        int conn = d_connectivityG1[idx];
        s_data[threadIdx.x] = conn;
    } else {
        s_data[threadIdx.x] = -INF;
    }

    __syncthreads();

    for(unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if(threadIdx.x < s) {
            s_data[threadIdx.x] = max(s_data[threadIdx.x], s_data[threadIdx.x + s]);
        }
        __syncthreads();
    }

    if(threadIdx.x == 0) {
        atomicMax(d_maxConnectivity, s_data[0]);
    }
}

__global__ void maxConnectivityFilterKernel(int* d_connectivityG1, int* d_maxConnectivity, int* d_is_good, int V) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx < V) {
        if(d_connectivityG1[idx] != *d_maxConnectivity) {
            d_is_good[idx] = 0;
        }
    }
}

__global__ void unorderedFilterKernel(int* d_is_good, int* d_V1Unordered, int* d_levelNodes, int V) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx >= V) {
        return;
    }

    if(d_levelNodes[idx]) {
        if(d_V1Unordered[idx] == 1) {
            d_is_good[idx] = 0;
        }
    } else {
        d_is_good[idx] = 0;
    }
}

__global__ void updateConnKernel(int* matrix, int V, int* connectivity, int node) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx < V) {
        if(matrix[node * V + idx] == 1) {
            atomicAdd(&connectivity[idx], 1);
        }
    }
}

// we need to update labelRarity so we cannot use constMem in order to avoid overhead
__global__ void maxRarityKernel(int V, int* d_labelRarity, int* d_nodesToLabel, int* d_maxRarity, int* d_is_good) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    extern __shared__ int s_data[];

    if(idx < V && d_is_good[idx]) {
        s_data[threadIdx.x] = d_labelRarity[d_nodesToLabel[idx]];
    } else {
        s_data[threadIdx.x] = INF;
    }

    __syncthreads();

    // parallel reduction: at each step, the block is halved and each thread computes the min of its value with the value of the other one at distance s in
    // the same block
    for(unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if(threadIdx.x < s) {
            s_data[threadIdx.x] = min(s_data[threadIdx.x], s_data[threadIdx.x + s]);   // At the end of the loop, the min value is in sdata[0]
        }
        __syncthreads();
    }

    // each thread 0 of each block computes the global min
    if(threadIdx.x == 0) {
        atomicMin(d_maxRarity, s_data[0]);
    }
}

__global__ void maxRarityFilterKernel(int V, int* d_labelRarity, int* d_nodesToLabel, int* d_maxRarity, int* d_is_good) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx < V) {
        if(d_labelRarity[d_nodesToLabel[idx]] != *d_maxRarity) {
            d_is_good[idx] = 0;
        }
    }
}

void processDepthGPU(Graph* d_g, int* order, int* order_index, int* d_connectivityG1, int* d_labelRarity, int* d_V1Unordered, int* d_levelNodes,
                    int* d_levelSize, int* d_is_good, int* d_maxRarity, int* d_maxDegree, int* d_maxConnectivity, int* d_nextNode, hipStream_t* streams,
                    Graph* h_g) {
    
    size_t size1 = d_g->numVertices * sizeof(int);
    size_t size2 = LABELS * sizeof(int);

    int h_levelSize;
    int* h_nodesToLabel = h_g->nodesToLabel;

    hipStream_t stream1 = streams[0];
    hipStream_t stream2 = streams[1];
    hipStream_t stream3 = streams[2];
    hipStream_t stream4 = streams[3];

    CUDA_CHECK_ERROR(hipMemcpyAsync(&h_levelSize, d_levelSize, sizeof(int), hipMemcpyDeviceToHost, stream1));

    // pinned memory for faster host-device communication
    int *h_maxRarity, *h_maxDegree, *h_maxConnectivity, *h_nextNode; 
    int* h_labelRarity, *h_V1Unordered;

    CUDA_CHECK_ERROR(hipHostMalloc((void**)&h_maxRarity, sizeof(int)));
    CUDA_CHECK_ERROR(hipHostMalloc((void**)&h_maxDegree, sizeof(int)));
    CUDA_CHECK_ERROR(hipHostMalloc((void**)&h_maxConnectivity, sizeof(int)));
    CUDA_CHECK_ERROR(hipHostMalloc((void**)&h_nextNode, sizeof(int)));

    CUDA_CHECK_ERROR(hipHostMalloc((void**)&h_labelRarity, size2));
    CUDA_CHECK_ERROR(hipHostMalloc((void**)&h_V1Unordered, size1));       

    *h_maxRarity = INF; *h_maxDegree = -INF; *h_maxConnectivity = -INF;

    int gridSize = (d_g->numVertices + blockSize - 1) / blockSize;
    size_t sharedMemSize = blockSize * sizeof(int);

    hipStreamSynchronize(stream1);

    while(h_levelSize > 0) {
        initArrayKernel<<<gridSize, blockSize, 0, stream1>>>(d_is_good, d_g->numVertices, 1);
        CUDA_CHECK_ERROR(hipMemcpyAsync(d_maxRarity, h_maxRarity, sizeof(int), hipMemcpyHostToDevice, stream2));
        CUDA_CHECK_ERROR(hipMemcpyAsync(d_maxDegree, h_maxDegree, sizeof(int), hipMemcpyHostToDevice, stream3));
        CUDA_CHECK_ERROR(hipMemcpyAsync(d_maxConnectivity, h_maxConnectivity, sizeof(int), hipMemcpyHostToDevice, stream4));

        hipStreamSynchronize(stream2);
        hipStreamSynchronize(stream3);
        hipStreamSynchronize(stream4);

        unorderedFilterKernel<<<gridSize, blockSize, 0, stream1>>>(d_is_good, d_V1Unordered, d_levelNodes, d_g->numVertices);
        maxConnectivityKernel<<<gridSize, blockSize, sharedMemSize, stream1>>>(d_connectivityG1, d_maxConnectivity, d_is_good, d_g->numVertices);
        maxConnectivityFilterKernel<<<gridSize, blockSize, 0, stream1>>>(d_connectivityG1, d_maxConnectivity, d_is_good, d_g->numVertices);
        maxDegreeKernel<<<gridSize, blockSize, sharedMemSize, stream1>>>(d_g->numVertices, d_g->degrees, d_maxDegree, d_is_good);   
        maxDegreeFilterKernel<<<gridSize, blockSize, 0, stream1>>>(d_g->numVertices, d_g->degrees, d_maxDegree, d_is_good);
        maxRarityKernel<<<gridSize, blockSize, sharedMemSize, stream1>>>(d_g->numVertices, d_labelRarity, d_g->nodesToLabel, d_maxRarity, d_is_good);
        maxRarityFilterKernel<<<gridSize, blockSize, 0, stream1>>>(d_g->numVertices, d_labelRarity, d_g->nodesToLabel, d_maxRarity, d_is_good);
        findNodeKernel<<<gridSize, blockSize, 0, stream1>>>(d_g->numVertices, d_is_good, d_nextNode);
        
        CUDA_CHECK_ERROR(hipMemcpyAsync(h_nextNode, d_nextNode, sizeof(int), hipMemcpyDeviceToHost, stream1));

        CUDA_CHECK_ERROR(hipMemcpyAsync(h_labelRarity, d_labelRarity, size2, hipMemcpyDeviceToHost, stream2));        
        CUDA_CHECK_ERROR(hipMemcpyAsync(h_V1Unordered, d_V1Unordered, size1, hipMemcpyDeviceToHost, stream3)); 
        
        hipStreamSynchronize(stream1);
        
        updateConnKernel<<<gridSize, blockSize, 0, stream1>>>(d_g->matrix, d_g->numVertices, d_connectivityG1, *h_nextNode);
        order[(*order_index)++] = *h_nextNode;
        h_levelSize--;

        hipStreamSynchronize(stream2);
        hipStreamSynchronize(stream3);

        h_labelRarity[h_nodesToLabel[*h_nextNode]]--;
        h_V1Unordered[*h_nextNode] = 1;

        CUDA_CHECK_ERROR(hipMemcpyAsync(d_labelRarity, h_labelRarity, size2, hipMemcpyHostToDevice, stream2));
        CUDA_CHECK_ERROR(hipMemcpyAsync(d_V1Unordered, h_V1Unordered, size1, hipMemcpyHostToDevice, stream3));

        hipStreamSynchronize(stream1);
        hipStreamSynchronize(stream2);
        hipStreamSynchronize(stream3);
    }

    hipHostFree(h_maxRarity);
    hipHostFree(h_maxDegree);
    hipHostFree(h_maxConnectivity);
    hipHostFree(h_nextNode);
    hipHostFree(h_labelRarity);
    hipHostFree(h_V1Unordered);
}

__global__ void findCoveredNeighborsKernel(int* matrix1, int* mapping1, int node, int* coveredNeighbors, int* coveredNeighborsSize,
                                            int numVertices) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx >= numVertices)
        return;

    if(matrix1[node * numVertices + idx] == 1 && mapping1[idx] != -1) {
        int index = atomicAdd(coveredNeighborsSize, 1);
        coveredNeighbors[index] = idx;
    }
}

__global__ void findCandidatesKernel(int g1_label, int maxSizeCandidates, int* g2_vertexList, int g1_degree, int* g2_degrees, int* T2_out, 
                                    int* mapping2, int* candidates, int* candidateSize, int g2_numVertices, int* commonNodes, int* g2_matrix, 
                                    int* g2_nodesToLabel, int offset, int useConstMem, int* coveredNeighbors, int* mapping1) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(constMemVar == 0) {  // constMemVar contains coveredNeighborsSize

        if(idx < maxSizeCandidates) {
            int vertex = g2_vertexList[idx];  // g2_labelToNodes[label]

            if(g2_degrees[vertex] == g1_degree && T2_out[vertex] == 1 && mapping2[vertex] == -1) {
                int index = atomicAdd(candidateSize, 1);
                candidates[index] = vertex;
            }
        }
    }
    else {

        if(idx < g2_numVertices) {
           commonNodes[idx] = 1;

            if(useConstMem) {
                
                for (int i = 0; i < constMemVar; i++) {
                    int nbrG1 = constMem[i];        // constMem contains coveredNeighbors with offset 0
                    int mappedG2 = constMem[offset + nbrG1]; // constMem contains mapping1 with offset degrees[node]
                    if (g2_matrix[mappedG2 * g2_numVertices + idx] == 0) {
                        commonNodes[idx] = 0;
                    }
                }

            } else {

                for (int i = 0; i < constMemVar; i++) {
                    int nbrG1 = coveredNeighbors[i];
                    int mappedG2 = mapping1[nbrG1];
                    if (g2_matrix[mappedG2 * g2_numVertices + idx] == 0) {
                        commonNodes[idx] = 0;
                    }
                }

            }

            if (commonNodes[idx] && mapping2[idx] != -1) {
                commonNodes[idx] = 0;
            }

            if (commonNodes[idx] && g2_degrees[idx] != g1_degree) {
                commonNodes[idx] = 0;
            }

            if (commonNodes[idx] && g2_nodesToLabel[idx] != g1_label) {
                commonNodes[idx] = 0;
            }

            if (commonNodes[idx] == 1) {
                int index = atomicAdd(candidateSize, 1);
                candidates[index] = idx;
            }
        }
    }
}

int* findCandidatesGPU(Graph* d_g1, Graph* d_g2, State* d_state, int node, int* sizeCandidates, Graph* h_g1, Graph* h_g2, hipStream_t* streams) {
    hipStream_t stream1 = streams[0];
    hipStream_t stream2 = streams[1];

    size_t neighSize = h_g1->degrees[node] * sizeof(int);
    size_t size2 = d_g2->numVertices * sizeof(int);
    size_t size1 = d_g1->numVertices * sizeof(int);

    int useConstMem = d_g1->numVertices + h_g1->degrees[node] < MAXCONSTMEM;

    // writes on constMem must be sequential
    if(useConstMem) {
        CUDA_CHECK_ERROR(hipMemcpyToSymbolAsync(HIP_SYMBOL(constMem), d_state->mapping1, size1, neighSize, hipMemcpyDeviceToDevice, stream2));    // neighSize is the offset
    }
    
    int* d_coveredNeighbors, *d_coveredNeighborsSize;
    
    CUDA_CHECK_ERROR(hipMalloc((void**)&d_coveredNeighbors, neighSize));
    CUDA_CHECK_ERROR(hipMalloc((void**)&d_coveredNeighborsSize, sizeof(int)));

    CUDA_CHECK_ERROR(hipMemsetAsync(d_coveredNeighborsSize, 0, sizeof(int), stream1));
    
    int gridSize = (d_g1->numVertices + blockSize - 1) / blockSize;

    findCoveredNeighborsKernel<<<gridSize, blockSize, 0, stream1>>>(d_g1->matrix, d_state->mapping1, node, d_coveredNeighbors, 
                                d_coveredNeighborsSize, d_g1->numVertices);

    int g1_label = h_g1->nodesToLabel[node];
    int g1_degree = h_g1->degrees[node];
    int maxSizeCandidates = h_g2->labelsCardinalities[g1_label];
    
    int* d_candidates, *d_candidateSize, *d_commonNodes;

    CUDA_CHECK_ERROR(hipMalloc((void**)&d_candidates, size2));
    CUDA_CHECK_ERROR(hipMalloc((void**)&d_candidateSize, sizeof(int)));
    CUDA_CHECK_ERROR(hipMalloc((void**)&d_commonNodes, size2));

    CUDA_CHECK_ERROR(hipMemsetAsync(d_candidateSize, 0, sizeof(int), stream2));
    
    gridSize = (d_g2->numVertices + blockSize - 1) / blockSize;
    
    hipStreamSynchronize(stream2); // transfer of constMem in the stream2 must be finished before the second write on constMem

    if(useConstMem) {
        CUDA_CHECK_ERROR(hipMemcpyToSymbolAsync(HIP_SYMBOL(constMem), d_coveredNeighbors, neighSize, 0, hipMemcpyDeviceToDevice, stream1));   // it is queued before the kernel call on stream1 so implicit synchronization
    }
    
    CUDA_CHECK_ERROR(hipMemcpyToSymbolAsync(HIP_SYMBOL(constMemVar), d_coveredNeighborsSize, sizeof(int), 0, hipMemcpyDeviceToDevice, stream1)); // constMemVar contains coveredNeighborsSize

    findCandidatesKernel<<<gridSize, blockSize, 0, stream1>>>(g1_label, maxSizeCandidates, d_g2->labelToNodes[g1_label], g1_degree, 
                            d_g2->degrees, d_state->T2_out, d_state->mapping2, d_candidates, d_candidateSize, d_g2->numVertices, 
                            d_commonNodes, d_g2->matrix, d_g2->nodesToLabel, h_g1->degrees[node], useConstMem, );

    int* candidates = (int*)malloc(maxSizeCandidates * sizeof(int));
    
    hipStreamSynchronize(stream1);

    CUDA_CHECK_ERROR(hipMemcpyAsync(candidates, d_candidates, maxSizeCandidates * sizeof(int), hipMemcpyDeviceToHost, stream2));
    CUDA_CHECK_ERROR(hipMemcpyAsync(sizeCandidates, d_candidateSize, sizeof(int), hipMemcpyDeviceToHost, stream1));

    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);

    hipFree(d_coveredNeighbors);
    hipFree(d_coveredNeighborsSize);

    hipFree(d_candidates);
    hipFree(d_candidateSize);
    hipFree(d_commonNodes);

    return candidates;
}


void vf2ppGPU(Graph* d_g1, Graph* d_g2, State* d_state, Graph* h_g1, Graph* h_g2, hipStream_t* streams) {
    if (!checkGraphPropertiesGPU(h_g1, h_g2, d_g1, d_g2, streams)) {
        return;
    }

    int* order = orderingGPU(d_g1, d_g2, streams, h_g1);

     //printf("Order:\t");
     //for(int i = 0; i < h_g1->numVertices; i++) {
     //   printf("%d ", order[i]);
     //}
     //printf("\n");

    int sizeCandidates = 0;
    int* candidates = findCandidatesGPU(d_g1, d_g2, d_state, order[0], &sizeCandidates, h_g1, h_g2, streams);

    StackNode* stack = createStack();
    Info* info = createInfo(candidates, sizeCandidates, order[0]);
    push(&stack, info);

    int matchingNode = 1;
    while (!isStackEmpty(stack)) {
        Info* info = peek(&stack);
        bool isMatch = false;

        // printInfo(info);

        for(int i = info->candidateIndex; i < info->sizeCandidates; i++) {
            int candidate = info->candidates[i];
            info->candidateIndex = i + 1;

            int ret = cutISOGPU(d_g1, d_g2, d_state, info->vertex, candidate, h_g1, h_g2, streams);

            // printf("CutISO: %d\n", ret);
            // printf("\n");

            if(!ret) {
                // printf("\nMatch %d -> %d\n", info->vertex, candidate);

                updateStateGPU(d_g1, d_g2, d_state, info->vertex, candidate, streams);

                if(matchingNode >= d_g1->numVertices) {
                    freeStack(stack);
                    free(order);
                    printf("Graphs are isomorphic\n");
                    hipStreamSynchronize(streams[0]);  // wait for updates on the state
                    hipStreamSynchronize(streams[1]);
                    return;
                }

                hipStreamSynchronize(streams[0]);
                hipStreamSynchronize(streams[1]);

                candidates = findCandidatesGPU(d_g1, d_g2, d_state, order[matchingNode], &sizeCandidates, h_g1, h_g2, streams);
                Info* info = createInfo(candidates, sizeCandidates, order[matchingNode]);
                push(&stack, info);
                matchingNode++;
                isMatch = true;
                break;
            }
        }

        // no more candidates
        if(!isMatch) {
            Info* tmp = pop(&stack);
            freeInfo(tmp);
            matchingNode--;

            // backtracking
            if(!isStackEmpty(stack)) {
                Info* prevInfo = peek(&stack);
                int candidate = prevInfo->candidates[prevInfo->candidateIndex - 1];
                restoreStateGPU(d_g1, d_g2, d_state, prevInfo->vertex, candidate, streams);
            }
        }
    }
    free(order);
    freeStack(stack);   
}

__global__ void findNeighborsKernel(int* matrix, int node, int* neighbors, int* size, int numVertices) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx < numVertices) {
        if(matrix[node * numVertices + idx] == 1) {
            int index = atomicAdd(size, 1);
            neighbors[index] = idx;
        }
    }
}

__global__ void checkLabelsKernel(int* neighbors1, int nbrSize1, int nbrSize2, int* labelsNbr, int numVertices,
    int* g1_nodesToLabel, int* d_result, int offset, int usage, int* neighbors2, int* g2_nodesToLabel) {   // idx is the id of the thread in the grid

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx < nbrSize1) {
        int nbr1 = neighbors1[idx];
        int labelNbr1 = g1_nodesToLabel[nbr1];
        bool found = false;

        if(usage) {

            for(int i = 0; i < nbrSize2; i++) {
                int nbr2 = constMem[i];     // constMem contains the neighbors2 with offset 0
                if(labelNbr1 == constMem[offset + nbr2]) {  // constMem contains the g2_nodesToLabel with offset degrees[node]
                    found = true;
                    labelsNbr[labelNbr1] = 1;
                    break;
                }
            }

        } else {

            for(int i = 0; i < *nbrSize2; i++) {
                int nbr2 = neighbors2[i];
                if(labelNbr1 == g2_nodesToLabel[nbr2]) {
                    found = true;
                    labelsNbr[labelNbr1] = 1;
                    break;
                }
            }
        }

        if(!found) {
            atomicExch(d_result, 0);   // d_result is initialized to 1 by default
        }
    }
}

__global__ void findNodesOfLabelKernel(int* neighbors, int* g_nodesToLabel, int label, int maxSize, int* size, int* nodes) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx < maxSize) {
        int vertex = neighbors[idx];

        if(g_nodesToLabel[vertex] == label) {
            int index = atomicAdd(size, 1);   // atomicAdd returns the index respect to global memory (so can't be used shared memory)
            nodes[index] = vertex;
        }
    }
}

__global__ void intersectionCountKernel(int* nodes, int* size, int* stateSet, int* count) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;    // idx is the id of the thread in the grid among all threads
    __shared__ int localCount;  // each block has own localCount variable (shared memory)

    if(threadIdx.x == 0) {   // only one thread in the block initializes the localCount
        localCount = 0;
    }

    __syncthreads();

    if(idx < *size) {
        int vertex = nodes[idx];

        if(stateSet[vertex] == 1) {
            atomicAdd(&localCount, 1);
        }
    }

    __syncthreads();

    if(threadIdx.x == 0) {  // only the thread with id 0 of each block updates the global size
        atomicAdd(count, localCount);
    }
}

bool cutISOGPU(Graph* d_g1, Graph* d_g2, State* d_state, int node1, int node2, Graph* h_g1, Graph* h_g2, hipStream_t* streams) {
    hipStream_t stream1 = streams[0];
    hipStream_t stream2 = streams[1];
    hipStream_t stream3 = streams[2];
    hipStream_t stream4 = streams[3];
    hipStream_t stream5 = streams[4];
    hipStream_t stream6 = streams[5];

    int nbrSize1 = h_g1->degrees[node1];
    int nbrSize2 = h_g2->degrees[node2];

    size_t nbrSize1_bytes = nbrSize1 * sizeof(int);
    size_t nbrSize2_bytes = nbrSize2 * sizeof(int);
    size_t size1 = d_g1->numVertices * sizeof(int);
    size_t size2 = d_g2->numVertices * sizeof(int);
    size_t labelSize = LABELS * sizeof(int);

    int useConstMem = d_g2->numVertices + nbrSize2 < MAXCONSTMEM;

    if(useConstMem) {
        CUDA_CHECK_ERROR(hipMemcpyToSymbolAsync(HIP_SYMBOL(constMem), d_g2->nodesToLabel, size2, nbrSize2_bytes, hipMemcpyDeviceToDevice, stream3)); // constMem contains g2_nodesToLabel
    }

    int* d_neighbors1, *d_neighbors2;
    int* d_nbrSize1, *d_nbrSize2;

    CUDA_CHECK_ERROR(hipMalloc((void**)&d_neighbors2, nbrSize2_bytes));
    CUDA_CHECK_ERROR(hipMalloc((void**)&d_nbrSize2, sizeof(int)));

    CUDA_CHECK_ERROR(hipMemsetAsync(d_nbrSize2, 0, sizeof(int), stream2));
   
    int gridSize = (d_g2->numVertices + blockSize - 1) / blockSize;

    findNeighborsKernel<<<gridSize, blockSize, 0, stream2>>>(d_g2->matrix, node2, d_neighbors2, d_nbrSize2, d_g2->numVertices);
    
    CUDA_CHECK_ERROR(hipMalloc((void**)&d_neighbors1, nbrSize1_bytes));
    CUDA_CHECK_ERROR(hipMalloc((void**)&d_nbrSize1, sizeof(int)));
    
    CUDA_CHECK_ERROR(hipMemsetAsync(d_nbrSize1, 0, sizeof(int), stream1));
    
    gridSize = (d_g1->numVertices + blockSize - 1) / blockSize;

    findNeighborsKernel<<<gridSize, blockSize, 0, stream1>>>(d_g1->matrix, node1, d_neighbors1, d_nbrSize1, d_g1->numVertices);

    int* d_labelsNbr;
    int* d_result;
    int result = 1;                               

    CUDA_CHECK_ERROR(hipMalloc((void**)&d_labelsNbr, labelSize));
    CUDA_CHECK_ERROR(hipMalloc((void**)&d_result, sizeof(int)));
    
    CUDA_CHECK_ERROR(hipMemsetAsync(d_labelsNbr, 0, labelSize, stream4));
    CUDA_CHECK_ERROR(hipMemcpyAsync(d_result, &result, sizeof(int), hipMemcpyHostToDevice, stream4));

    gridSize = (nbrSize1 + blockSize - 1) / blockSize;

    hipStreamSynchronize(stream3); // the first write on constMem must be finished before the second write on constMem
    
    if(useConstMem) {
        CUDA_CHECK_ERROR(hipMemcpyToSymbolAsync(HIP_SYMBOL(constMem), d_neighbors2, nbrSize2_bytes, 0, hipMemcpyDeviceToDevice, stream2)); // constMem contains neighbors2. it is queued after findNeighborsKernel on stream2 
    }

    hipStreamSynchronize(stream4);
    hipStreamSynchronize(stream2);

    checkLabelsKernel<<<gridSize, blockSize, 0, stream1>>>(d_neighbors1, nbrSize1, nbrSize2, d_labelsNbr, d_g1->numVertices, 
                        d_g1->nodesToLabel, d_result, nbrSize2, useConstMem, d_neighbors2, d_g2->nodesToLabel);

    int* labelsNbr = (int*)malloc(labelSize);
    CUDA_CHECK_ERROR(hipMemcpyAsync(&result, d_result, sizeof(int), hipMemcpyDeviceToHost, stream1));
    CUDA_CHECK_ERROR(hipMemcpyAsync(labelsNbr, d_labelsNbr, labelSize, hipMemcpyDeviceToHost, stream1));

    hipStreamSynchronize(stream1);

    if(result == 0) {
        hipFree(d_neighbors1);
        hipFree(d_neighbors2);
        hipFree(d_nbrSize1);
        hipFree(d_nbrSize2);
        hipFree(d_labelsNbr);
        hipFree(d_result);
        free(labelsNbr);
        return true;
    }

    // pinned memory for faster host-device communication
    int *h_size1, *h_size2;
    int *h_count1, *h_count2, *h_count3, *h_count4;
    int *d_count1, *d_count2, *d_count3, *d_count4;

    CUDA_CHECK_ERROR(hipHostMalloc((void**)&h_size1, sizeof(int)));
    CUDA_CHECK_ERROR(hipHostMalloc((void**)&h_size2, sizeof(int)));
    CUDA_CHECK_ERROR(hipHostMalloc((void**)&h_count1, sizeof(int)));
    CUDA_CHECK_ERROR(hipHostMalloc((void**)&h_count2, sizeof(int)));
    CUDA_CHECK_ERROR(hipHostMalloc((void**)&h_count3, sizeof(int)));
    CUDA_CHECK_ERROR(hipHostMalloc((void**)&h_count4, sizeof(int)));

    CUDA_CHECK_ERROR(hipMalloc((void**)&d_count1, sizeof(int)));
    CUDA_CHECK_ERROR(hipMalloc((void**)&d_count2, sizeof(int)));
    CUDA_CHECK_ERROR(hipMalloc((void**)&d_count3, sizeof(int)));
    CUDA_CHECK_ERROR(hipMalloc((void**)&d_count4, sizeof(int)));

    int *d_nodes_g1, *d_size_g1;
    int *d_nodes_g2, *d_size_g2;

    CUDA_CHECK_ERROR(hipMalloc((void**)&d_nodes_g1, nbrSize1_bytes));
    CUDA_CHECK_ERROR(hipMalloc((void**)&d_size_g1, sizeof(int)));
    CUDA_CHECK_ERROR(hipMalloc((void**)&d_nodes_g2, nbrSize2_bytes));
    CUDA_CHECK_ERROR(hipMalloc((void**)&d_size_g2, sizeof(int)));

    int gridSize1 = (nbrSize1 + blockSize - 1) / blockSize;
    int gridSize2 = (nbrSize2 + blockSize - 1) / blockSize;
   
    bool ret = false;
    for(int label = 0; label < LABELS; label++) {
        if(labelsNbr[label] == 1) {

            CUDA_CHECK_ERROR(hipMemsetAsync(d_size_g1, 0, sizeof(int), stream1));
            CUDA_CHECK_ERROR(hipMemsetAsync(d_size_g2, 0, sizeof(int), stream2));

            findNodesOfLabelKernel<<<gridSize1, blockSize, 0, stream1>>>(d_neighbors1, d_g1->nodesToLabel, label, nbrSize1, d_size_g1, d_nodes_g1);
            findNodesOfLabelKernel<<<gridSize2, blockSize, 0, stream2>>>(d_neighbors2, d_g2->nodesToLabel, label, nbrSize2, d_size_g2, d_nodes_g2);

            CUDA_CHECK_ERROR(hipMemcpyAsync(h_size1, d_size_g1, sizeof(int), hipMemcpyDeviceToHost, stream1));
            CUDA_CHECK_ERROR(hipMemcpyAsync(h_size2, d_size_g2, sizeof(int), hipMemcpyDeviceToHost, stream2));

            CUDA_CHECK_ERROR(hipMemsetAsync(d_count1, 0, sizeof(int), stream3));
            CUDA_CHECK_ERROR(hipMemsetAsync(d_count2, 0, sizeof(int), stream4));
            CUDA_CHECK_ERROR(hipMemsetAsync(d_count3, 0, sizeof(int), stream5));
            CUDA_CHECK_ERROR(hipMemsetAsync(d_count4, 0, sizeof(int), stream6));

            hipStreamSynchronize(stream1);
            hipStreamSynchronize(stream2);

            gridSize = (*h_size1 + blockSize - 1) / blockSize;
            intersectionCountKernel<<<gridSize, blockSize, 0, stream3>>>(d_nodes_g1, d_size_g1, d_state->T1, d_count1);
            intersectionCountKernel<<<gridSize, blockSize, 0, stream5>>>(d_nodes_g1, d_size_g1, d_state->T1_out, d_count3);

            CUDA_CHECK_ERROR(hipMemcpyAsync(h_count1, d_count1, sizeof(int), hipMemcpyDeviceToHost, stream3));
            CUDA_CHECK_ERROR(hipMemcpyAsync(h_count3, d_count3, sizeof(int), hipMemcpyDeviceToHost, stream5));
            
            gridSize = (*h_size2 + blockSize - 1) / blockSize;
            intersectionCountKernel<<<gridSize, blockSize, 0, stream4>>>(d_nodes_g2, d_size_g2, d_state->T2, d_count2);
            intersectionCountKernel<<<gridSize, blockSize, 0, stream6>>>(d_nodes_g2, d_size_g2, d_state->T2_out, d_count4);

            CUDA_CHECK_ERROR(hipMemcpyAsync(h_count2, d_count2, sizeof(int), hipMemcpyDeviceToHost, stream4));
            CUDA_CHECK_ERROR(hipMemcpyAsync(h_count4, d_count4, sizeof(int), hipMemcpyDeviceToHost, stream6));
           
            hipStreamSynchronize(stream3);
            hipStreamSynchronize(stream4);
            hipStreamSynchronize(stream5);
            hipStreamSynchronize(stream6);

            if(*h_count1 != *h_count2 || *h_count3 != *h_count4) {
                ret = true;
                break;
            }
        }
    }

    hipFree(d_neighbors1);
    hipFree(d_neighbors2);
    hipFree(d_nbrSize1);
    hipFree(d_nbrSize2);
    hipFree(d_labelsNbr);
    hipFree(d_result);
    free(labelsNbr);

    hipHostFree(h_size1);
    hipHostFree(h_size2);
    hipHostFree(h_count1);
    hipHostFree(h_count2);
    hipHostFree(h_count3);
    hipHostFree(h_count4);
    
    hipFree(d_count1);
    hipFree(d_count2);
    hipFree(d_count3);
    hipFree(d_count4);

    hipFree(d_nodes_g1);
    hipFree(d_size_g1);
    hipFree(d_nodes_g2);
    hipFree(d_size_g2);

    return ret;
}

/***** STACK FUNCTIONS *****/
Info* createInfo(int* candidates, int sizeCandidates, int vertex) {
    Info* info = (Info*)malloc(sizeof(Info));
    if (info == NULL) {
        printf("Error allocating memory in createInfo\n");
        exit(EXIT_FAILURE);
    }
    info->vertex = vertex;
    info->candidates = (int*)realloc(candidates, sizeCandidates * sizeof(int));
    info->sizeCandidates = sizeCandidates;
    info->candidateIndex = 0;
    return info;
}

StackNode* createStackNode(Info* info) {
    StackNode* node = (StackNode*)malloc(sizeof(StackNode));
    if (node == NULL) {
        printf("Error allocating memory in createStackNode\n");
        exit(EXIT_FAILURE);
    }
    node->info = info;
    node->next = NULL;
    return node;
}

void push(StackNode** top, Info* info) {
    StackNode* node = createStackNode(info);
    node->next = *top;
    *top = node;
}

Info* pop(StackNode** top) {
    if (isStackEmpty(*top)) {
        printf("Stack is empty, cannot pop\n");
        return NULL;
    }
    StackNode* node = *top;
    Info* info = node->info;
    *top = node->next;
    free(node);
    return info;
}

bool isStackEmpty(StackNode* top) {
    return top == NULL;
}

void freeStack(StackNode* top) {
    while (!isStackEmpty(top)) {
        StackNode* node = top;
        top = top->next;
        freeInfo(node->info);
        free(node);
    }
}

void printStack(StackNode* top) {
    StackNode* current = top;
    while (current != NULL) {
        printInfo(current->info);
        current = current->next;
    }
}

void printInfo(Info* info) {
    printf("\nVertex: %d\n", info->vertex);
    printf("Index seen: %d\n", info->candidateIndex);
    printf("Candidates: ");
    for (int i = 0; i < info->sizeCandidates; i++) {
        printf("%d ", info->candidates[i]);
    }
    printf("\n");
}

void freeInfo(Info* info) {
    free(info->candidates);
    free(info);
}

StackNode* createStack() {
    return NULL;
}

Info* peek(StackNode** top) {
    return (*top)->info;
}